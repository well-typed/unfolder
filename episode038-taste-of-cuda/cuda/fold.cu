#include "hip/hip_runtime.h"
extern "C" {
    #include "ep38-cuda.h"
}

#define FULL_WARP 0xffffffff

__global__ void ep38_fold_add_kernel(float* inout_d) {
    unsigned int i = threadIdx.x;

    for (unsigned int stride = blockDim.x; stride > 16; stride /= 2) {
        if(i < stride) {
            inout_d[i] += inout_d[i + stride];
        }
        __syncthreads();
    }
}

__global__ void ep38_fold_add_warp(float* inout_d, unsigned int n) {
    unsigned int i = threadIdx.x;
    float val = 0;

    if(i < n) {
        val = inout_d[i];
    }

    for (int stride = 16; stride > 0; stride /= 2) {
        val += __shfl_down_sync(FULL_WARP, val, stride);
    }

    if(i == 0) {
        inout_d[i] = val;
    }
}

extern "C" {
    float ep38_fold_add(float* inp_h, unsigned int n) {
        float* inout_d = NULL;
        unsigned int numBytes = n * sizeof(float);
        hipMalloc((void**) &inout_d, numBytes);
        hipMemcpy(inout_d, inp_h, numBytes, hipMemcpyHostToDevice);

        if(n <= 32) {
            ep38_fold_add_warp <<<1,32>>> (inout_d, n);
        } else {
            // https://jameshfisher.com/2018/03/30/round-up-power-2/
            unsigned int num_threads = 1 << (31 - __builtin_clz(n-1));
            ep38_fold_add_kernel <<<1,num_threads>>> (inout_d);
            ep38_fold_add_warp <<<1,32>>> (inout_d, n);
        }

        float result_h;
        hipMemcpy(&result_h, inout_d, sizeof(float), hipMemcpyDeviceToHost);
        hipFree(inout_d);
        return result_h;
    }
}