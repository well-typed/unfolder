#include "hip/hip_runtime.h"
extern "C" {
    #include "ep38-cuda.h"
}

#define SWAP(x, y) { typeof(x) SWAP = x; x = y; y = SWAP; }

__global__ void ep38_scan_add_kernel(float* inout_d, unsigned int n) {
    __shared__ float tmp1[1024];
    __shared__ float tmp2[1024];

    unsigned int i = threadIdx.x;

    float* r = tmp1;
    float* w = tmp2;

    r[i] = inout_d[i];
    __syncthreads();

    for(unsigned int stride = 1; stride < n; stride *= 2) {
        // After `k` iterations, w[i] contains the sum of the `2^k` elements
        // of `r`, leading up to (including) r[i].
        if(i >= stride) {
            w[i] = r[i] + r[i - stride];
        } else {
            w[i] = r[i];
        }
        __syncthreads();

        SWAP(r, w);
    }

    inout_d[i] = r[i];
}

extern "C" {
    void ep38_scan_add(float* inp_h, float* out_h, unsigned int n) {
        float* inout_d = NULL;
        unsigned int numBytes = n * sizeof(float);
        hipMalloc((void**) &inout_d, numBytes);
        hipMemcpy(inout_d, inp_h, numBytes, hipMemcpyHostToDevice);

        ep38_scan_add_kernel <<<1,n>>> (inout_d, n);

        hipMemcpy(out_h, inout_d, numBytes, hipMemcpyDeviceToHost);
        hipFree(inout_d);
    }
}