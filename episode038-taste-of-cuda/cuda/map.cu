#include "hip/hip_runtime.h"
extern "C" {
    #include "ep38-cuda.h"
}

__global__ void ep38_map_add_kernel(float x, float* inout_d) {
    unsigned int i = threadIdx.x;
    inout_d[i] += x;
}

extern "C" {
    void ep38_map_add(float x, float* inp_h, float* out_h, unsigned int n) {
        float* inout_d = NULL;
        unsigned int numBytes = n * sizeof(float);
        hipMalloc((void**) &inout_d, numBytes);
        hipMemcpy(inout_d, inp_h, numBytes, hipMemcpyHostToDevice);

        ep38_map_add_kernel <<<1,n>>> (x, inout_d);

        hipMemcpy(out_h, inout_d, numBytes, hipMemcpyDeviceToHost);
        hipFree(inout_d);
    }
}